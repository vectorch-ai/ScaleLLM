#include "hip/hip_runtime.h"
// Adapted from
// https://github.com/NVIDIA/TransformerEngine/blob/main/transformer_engine/common/permutation/permutation.cu
#include <ATen/cuda/HIPContext.h>
#include <torch/torch.h>

#include <hipcub/hipcub.hpp>
#include <cute/config.hpp>
#include <cute/numeric/numeric_types.hpp>

// clang-format off
// for exmple: n_tokens = 4, n_experts = 4, topk = 2, block_size=2
// f_idx: idx in flatten indices
// p_idx: idx in permuted tokens
// k_idx: topk idx
// t_idx: token idx
// row_id_map: [topk, n_tokens] => idx in permuted tokens
//  _______________________________________________________________________________________________________
// |                         |                         |                         |                         |
// |    Steps                |        routing_map      |   row_id_map -> cu_sum  |   row_id_map -> p_idx   |
// |                         |   [n_tokens, n_experts] |  [n_experts, n_tokens]  |  [n_experts, n_tokens]  |
// |_________________________|_________________________|_________________________|_________________________|
// |                         |                         |                         |                         |
// |                         |           e_idx         |           t_idx         |           t_idx         |
// |      top0, top1         |          0 1 2 3        |          0 1 | 2 3      |          0 1 | 2 3      |
// | t0 -> [e2, e1]          |    t0  | 0 1 1 0 |      |    e0  | x 1 | 1 x |    |    e0  | x 0 | 1 x |    |
// | t1 -> [e1, e0]          |    t1  | 1 1 0 0 |      |    e1  | 1 2 | 1 x |    |    e1  | 2 3 | 4 x |    |
// | t2 -> [e0, e1]          |    t2  | 1 1 0 0 |      |    e2  | 1 x | x 1 |    |    e2  | 5 x | x 6 |    |
// | t3 -> [e2, e3]          |    t3  | 0 0 1 1 |      |    e3  | x x | x 1 |    |    e3  | x x | x 7 |    |
// |_________________________|_________________________|_________________________|_________________________|
// |                         |                         |        block_sum        |      block_cu_sum       |
// |                         |                         |    b0     1  |  1       |  c_b0     0  |  1       |
// |                         |                         |    b1     2  |  1       |  c_b1     2  |  4       |
// |                         |                         |    b2     1  |  1       |  c_b2     5  |  6       |
// |                         |                         |    b3     0  |  1       |  c_b3     7  |  7       |
// |_________________________|_________________________|_________________________|_________________________|
// clang-format on

namespace llm::kernel::moe {

namespace {
template <typename T>
inline T* data_ptr(torch::Tensor& t) {
  return reinterpret_cast<T*>(t.data_ptr());
}

template <typename T>
inline const T* const_data_ptr(torch::Tensor& t) {
  return reinterpret_cast<const T*>(t.const_data_ptr());
}

// (grid, block) = (n_experts, min(n_blocks, 1024))
template <int BLOCK_SIZE>
__global__ void block_sum_kernel(
    const int* routing_map,  // [n_tokens, n_experts]
    int* row_id_map,         // [n_experts, n_tokens]
    int* block_sum,          // [n_experts, n_blocks]
    const int n_tokens,
    const int n_experts,
    const int n_blocks) {
  // expert idx
  const int e_idx = blockIdx.x;
  // start token idx
  const int tid = threadIdx.x;

  // process each token block
  for (int b = tid; b < n_blocks; b += blockDim.x) {
    // block start token idx
    const int t_base = b * BLOCK_SIZE;
    int sum = 0;
    // process each token in the block
    for (int i = 0; i < BLOCK_SIZE; ++i) {
      const int t_idx = t_base + i;
      if (t_idx < n_tokens) {
        // routing_map: [n_tokens, n_experts]
        const auto val = routing_map[(t_idx * n_experts) + e_idx];
        // row_id_map: [n_experts, n_tokens]
        row_id_map[(e_idx * n_tokens) + t_idx] = val ? ++sum : 0;
      } else {
        // out of range
        row_id_map[(e_idx * n_tokens) + t_idx] = 0;
      }
    }
    // block_sum: [n_experts, n_blocks]
    block_sum[(e_idx * n_blocks) + b] = sum;
  }
}

// (grid, block) = (n_experts, min(n_blocks, 1024))
template <int BLOCK_SIZE>
__global__ void row_id_map_kernel(
    const int* block_sum,  // [n_experts, n_blocks]
    int* row_id_map,       // [n_experts, n_tokens]
    const int n_tokens,
    const int n_experts,
    const int n_blocks) {
  // expert idx
  const int e_idx = blockIdx.x;
  // start token idx
  const int tid = threadIdx.x;
  const int total_blocks = n_experts * n_blocks;
  // process each token block
  for (int b = tid; b < n_blocks; b += blockDim.x) {
    const int g_b = n_blocks * e_idx + b;
    int cu_sum = 0;
    for (int i = 0; i < g_b; ++i) {
      cu_sum += block_sum[i];
    }

    // block start token idx
    const int t_base = b * BLOCK_SIZE;
    int sum = 0;
    // process each token in the block
    for (int i = 0; i < BLOCK_SIZE; ++i) {
      const int t_idx = t_base + i;
      // row_id_map: [n_experts, n_tokens]
      const int idx = (e_idx * n_tokens) + t_idx;
      if (t_idx < n_tokens) {
        const auto val = row_id_map[idx];
        row_id_map[idx] = val ? cu_sum + val - 1 : -1;
      } else {
        row_id_map[idx] = -1;
      }
    }
  }
}

template <typename T>
__global__ void permute_kernel(
    const T* tokens,        // [n_tokens, dim]
    T* permuted_tokens,     // [n_permuted_tokens, dim]
    const int* row_id_map,  // [n_experts, n_tokens] => p_idx
    const int n_tokens,
    const int n_experts,
    const int dim) {
  // one block corresponds to one token
  const int t_idx = blockIdx.x;
  const int tid = threadIdx.x;

  // frag for load/store
  float4 frag_ls;

  static constexpr int kFragSize = 16 / sizeof(T);
  // tokens: [n_tokens, dim]
  const T* token_base = tokens + t_idx * dim;
  for (int i = tid * kFragSize; i < dim; i += blockDim.x * kFragSize) {
    // load fragment into frag_ls (float4)
    frag_ls = __ldlu(reinterpret_cast<const float4*>(token_base + i));

    // broadcast to all experts
    for (int e_idx = 0; e_idx < n_experts; ++e_idx) {
      // row_id_map: [n_experts, n_tokens] => idx in permuted tokens
      const int p_idx = row_id_map[(e_idx * n_tokens) + t_idx];
      if (p_idx != -1) {
        // store back to permuted_tokens: [n_permuted_tokens, dim]
        T* permuted_token_base = permuted_tokens + p_idx * dim;
        *reinterpret_cast<float4*>(permuted_token_base + i) = frag_ls;
      }
    }
  }
}

template <typename T>
__global__ void unpermute_kernel(
    const T* permuted_tokens,  // [n_permuted_tokens, dim]
    T* tokens,                 // [n_tokens, dim]
    const int* row_id_map,  // [n_experts, n_tokens] => idx in permuted tokens
    const T* probs,         // [n_tokens, n_experts]
    const int n_tokens,
    const int n_experts,
    const int dim) {
  extern __shared__ int8_t s_mem[];
  // [topk] probs for the token
  T* s_probs = reinterpret_cast<T*>(s_mem);

  // each block corresponds to one token
  const int t_idx = blockIdx.x;
  const int tid = threadIdx.x;

  // load prob into shared memory for the token
  for (int i = tid; i < n_experts; i += blockDim.x) {
    s_probs[i] = probs[(t_idx * n_experts) + i];
  }
  __syncthreads();

  // float4 for load and store
  float4 frag_ls;
  T* frag_ls_ptr = reinterpret_cast<T*>(&frag_ls);

  static constexpr int kFragSize = 16 / sizeof(T);
  for (int i = tid * kFragSize; i < dim; i += blockDim.x * kFragSize) {
    T frag_sum[kFragSize] = {T(0.0f)};

    // sum over experts
    for (int e_idx = 0; e_idx < n_experts; ++e_idx) {
      // row_id_map: [n_experts, n_tokens] => idx in permuted tokens
      const int p_idx = row_id_map[(e_idx * n_tokens) + t_idx];
      if (p_idx != -1) {
        const T* permuted_token_base = permuted_tokens + p_idx * dim;
        // load fragment into frag_ls (float4)
        frag_ls =
            __ldlu(reinterpret_cast<const float4*>(permuted_token_base + i));

        // apply probs & sum
        const auto prob = s_probs[e_idx];
        CUTE_UNROLL
        for (int d = 0; d < kFragSize; ++d) {
          frag_sum[d] += (frag_ls_ptr[d] * prob);
        }
      }
    }

    // store back to tokens: [n_tokens, dim]
    T* token_base = tokens + t_idx * dim;
    *reinterpret_cast<float4*>(token_base + i) =
        *reinterpret_cast<float4*>(frag_sum);
  }
}

template <typename T>
void launch_permute_kernel(
    const T* tokens,           // [n_tokens, dim]
    T* permuted_tokens,        // [n_permuted_tokens, dim]
    const int* sorted_row_id,  // [n_permuted_tokens] -> flattened index
    int* row_id_map,           // [topk, n_tokens] -> idx in permuted tokens
    const int n_tokens,
    const int topk,
    const int dim,
    hipStream_t stream) {
  const int n_permuted_tokens = n_tokens * topk;
  int threads = 64;
  int blocks = (n_permuted_tokens + threads - 1) / threads;
  permute_row_id_map<<<blocks, threads, 0, stream>>>(
      sorted_row_id, row_id_map, n_tokens, topk);

  // use 128-bit load/store
  constexpr int kFragSize = 16 / sizeof(T);
  // assert(dim % kFragSize == 0);

  // one block per source token
  blocks = n_tokens;
  threads = std::min(dim / kFragSize, 1024);
  permute_kernel<T><<<blocks, threads, 0, stream>>>(
      tokens, permuted_tokens, row_id_map, n_tokens, topk, dim);
}

template <typename T>
void launch_unpermute_kernel(
    const T* permuted_tokens,  // [n_permuted_tokens, dim]
    T* tokens,                 // [n_tokens, dim]
    int* row_id_map,           // [topk, n_tokens] => dst row
    const T* prob,             // [n_tokens, topk]
    const int n_tokens,
    const int topk,
    const int dim,
    hipStream_t stream) {
  // use 128-bit load/store
  constexpr int kFragSize = 16 / sizeof(T);
  // assert(dim % kFragSize == 0);

  // each block corresponds to one token
  int blocks = n_tokens;
  // up to 1024 threads per block
  int threads = std::min(dim / kFragSize, 1024);
  size_t smem_bytes = topk * sizeof(T);

  // unpermute_topK fwd
  unpermute_kernel<T><<<blocks, threads, smem_bytes, stream>>>(
      permuted_tokens, tokens, row_id_map, prob, n_tokens, topk, dim);
}

}  // namespace

std::tuple<torch::Tensor, torch::Tensor> permute_with_mask_map(
    torch::Tensor tokens,       // [n_tokens, dim]
    torch::Tensor routing_map,  // [n_tokens, n_experts] bool/int tensor
    int64_t topk) {
  const auto n_tokens = tokens.size(0);
  const auto n_experts = routing_map.size(1);

  const auto options = tokens.options();
  const auto int32_options = options.dtype(torch::kInt32);

  auto row_id_map = torch::empty({n_experts, n_tokens}, int32_options);

  // step1: transpose routing_map to [n_experts, n_tokens] and calculate block
  // sum for each expert
  // launch_block_sum_kernel;

  // step2: calculate index in permuted tokens for each token
  // launch_row_id_kernel;

  const auto n_permuted_tokens = n_tokens * topk;

  const auto dim = tokens.size(1);
  const auto type = tokens.scalar_type();

  auto permuted_tokens = torch::empty({n_permuted_tokens, dim}, options);
  //   auto row_id_map = torch::empty(
  //       {n_tokens * topk}, torch::dtype(torch::kInt32).device(torch::kCUDA));

  //   auto* stream = at::cuda::getCurrentCUDAStream().stream();

  // #define LAUNCH_PERMUTE_KERNEL(DType)                             \
//   launch_permute_kernel<DType>(const_data_ptr<DType>(tokens),    \
//                                data_ptr<DType>(permuted_tokens), \
//                                sorted_row_id_ptr,                \
//                                row_id_map.data_ptr<int>(),       \
//                                n_tokens,                         \
//                                topk,                             \
//                                dim,                              \
//                                stream);

  //   switch (type) {
  //     case torch::ScalarType::Float: {
  //       LAUNCH_PERMUTE_KERNEL(float);
  //       break;
  //     }
  //     case torch::ScalarType::Half: {
  //       LAUNCH_PERMUTE_KERNEL(cute::half_t);
  //       break;
  //     }
  //     case torch::ScalarType::BFloat16: {
  //       LAUNCH_PERMUTE_KERNEL(cute::bfloat16_t);
  //       break;
  //     }
  //     default:
  //       CHECK(false) << "Unsupported tensor type: " << type;
  //   }

  return {permuted_tokens, row_id_map};
}

torch::Tensor unpermute_with_mask_map(
    torch::Tensor permuted_tokens,  // [n_permuted_tokens, dim]
    torch::Tensor row_id_map,       // [n_experts, n_tokens] => dst row
    torch::Tensor probs,            // [n_tokens, topk]
    int64_t n_tokens,
    int64_t topk) {
  const auto dim = permuted_tokens.size(1);
  const auto type = permuted_tokens.scalar_type();

  // [n_tokens, dim]
  auto tokens = torch::empty(
      {n_tokens, dim},
      torch::dtype(type).device(torch::kCUDA).requires_grad(false));

  auto* stream = at::cuda::getCurrentCUDAStream().stream();

#define LAUNCH_UNPERMUTE_KERNEL(DType)                                   \
  launch_unpermute_kernel<DType>(const_data_ptr<DType>(permuted_tokens), \
                                 data_ptr<DType>(tokens),                \
                                 row_id_map.data_ptr<int>(),             \
                                 const_data_ptr<DType>(probs),           \
                                 n_tokens,                               \
                                 topk,                                   \
                                 dim,                                    \
                                 stream);

  switch (type) {
    case torch::ScalarType::Float: {
      LAUNCH_UNPERMUTE_KERNEL(float);
      break;
    }
    case torch::ScalarType::Half: {
      LAUNCH_UNPERMUTE_KERNEL(cute::half_t);
      break;
    }
    case torch::ScalarType::BFloat16: {
      LAUNCH_UNPERMUTE_KERNEL(cute::bfloat16_t);
      break;
    }
    default:
      CHECK(false) << "Unsupported tensor type: " << type;
  }

  return tokens;
}

}  // namespace llm::kernel::moe
