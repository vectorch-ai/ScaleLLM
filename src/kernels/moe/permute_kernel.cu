#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
// #include <hip/hip_bf16.h>
#include <cutlass/arch/memory.h>
#include <cutlass/array.h>
#include <torch/torch.h>

#include <hipcub/hipcub.hpp>

// clang-format off
// for exmple: n_tokens = 2, n_experts = 8, topk = 2
//  ____________________________________________________________________________________________________________________________
// |                 |     flatten indices         |        sort flatten indices          |           row_id_map                |
// |    Steps        |   sort by (tokens, topk)    |        by (experts, tokens)          |     sort by (topk, tokens)          |
// |_________________|_____________________________|______________________________________|_____________________________________|
// |                 |    [n_tokens * topk]        |     [n_tokens * topk] => f_idx       |      [topk, n_tokens] => p_idx      |
// |     Dim         |                             |   f_idx: idx in flatten indices      |    p_idx: idx in permuted tokens    |
// |_________________|_____________________________|______________________________________|_____________________________________|
// |                 |                             |                                      |                                     |
// |      top0, top1 |   f_idx: | 0 | 1 | 2 | 3 |  |   p_idx: |  0  |  1  |  2  |  3  |   |     idx: |  0  |  1  |  2  |  3  |  |
// | t0 -> [e2, e1]  | experts: | 2 | 1 | 2 | 5 |  |   f_idx: |  1  |  0  |  2  |  3  |   |   p_idx: |  1  |  2  |  0  |  3  |  |
// | t1 -> [e2, e5]  |  tokens: |   t0  |   t1  |  |  tokens: |  t0 |  t0 |  t1 |  t1 |   |   f_idx: |  0  |  2  |  1  |  3  |  |
// |                 |                             | experts: |  e1 |     e2    |  e5 |   | experts: |  e2 |  e2 |  e1 |  e5 |  |
// |                 |                             |                                      |  tokens: |  t0 |  t1 |  t0 |  t1 |  |
// |                 |                             |                                      |    topk: |    top0   |    top1   |  |
// |_________________|_____________________________|______________________________________|_____________________________________|
// clang-format on

namespace llm::kernel::moe {

namespace {
template <typename T>
inline T* get_ptr(torch::Tensor& t) {
  return reinterpret_cast<T*>(t.data_ptr());
}

// build a row_id_map that maps [topk, n_tokens] to the index in permuted tokens
__global__ void permute_row_id_map(
    const int* sorted_row_id,  // [n_permuted_tokens]
    int* row_id_map,           // [topk, n_tokens]
    const int n_tokens,
    const int topk) {
  // row_id_map[num_topK][num_rows]
  const int bid = blockIdx.x;
  const int tid = threadIdx.x;

  // idx in permuted tokens
  const int p_idx = bid * blockDim.x + tid;
  const int n_permuted_tokens = n_tokens * topk;

  if (p_idx >= n_permuted_tokens) {
    return;
  }

  // idx in flattened indices
  const int f_idx = sorted_row_id[p_idx];
  // token idx: each token has topk experts in flattened indices
  const int token_idx = f_idx / topk;
  // topk idx: idx in topk experts for the token
  const int topk_idx = f_idx % topk;

  // row_id_map: [topk, n_tokens] => idx in permuted tokens
  row_id_map[(topk_idx * n_tokens) + token_idx] = p_idx;
}

template <typename T,
          int kFragSize,
          int kTopK>
__global__ void permute_kernel(
    const T* tokens,        // [n_tokens, dim]
    T* permuted_tokens,     // [n_permuted_tokens, dim]
    const int* row_id_map,  // [topk, n_tokens] => dst row
    const int n_tokens,
    const int topk,
    const int dim) {
  using Fragment = cutlass::Array<T, kFragSize>;

  // one block corresponds to one token
  const int token_idx = blockIdx.x;
  const int tid = threadIdx.x;

  Fragment frag;

  // tokens: [n_tokens, dim]
  const T* token_base = tokens + token_idx * dim;
  for (int i = tid * kFragSize; i < dim; i += blockDim.x * kFragSize) {
    // read one fragment
    cutlass::arch::global_load<Fragment,
                               sizeof(Fragment),
                               cutlass::arch::CacheOperation::LastUse>(
        frag, (token_base + i), true);

    int src_idx = token_idx;
    for (int k = 0; k < kTopK; k++) {
      if (k == topk) {
        break;
      }

      // row_id_map: [topk, n_tokens] => idx in permuted tokens
      const int dest_idx = row_id_map[src_idx];
      // move to next k
      src_idx += n_tokens;

      // permuted_tokens: [n_permuted_tokens, dim]
      T* permuted_token_base = permuted_tokens + dest_idx * dim;
      // use 128-bit copy
      *(float4*)(permuted_token_base + i) = *(float4*)(frag.data());
    }
  }
}

template <typename T, int kFragSize>
__global__ void unpermute_kernel(
    const T* permuted_tokens,  // [n_permuted_tokens, dim]
    T* tokens,                 // [n_tokens, dim]
    const int* row_id_map,     // [topk, n_tokens] => idx in permuted tokens
    const T* probs,            // [n_tokens, topk]
    const int n_tokens,
    const int topk,
    const int dim) {
  extern __shared__ int8_t s_mem[];
  // [topk] probs for the token
  T* s_probs = reinterpret_cast<T*>(s_mem);

  using Fragment = cutlass::Array<T, kFragSize>;

  // each block corresponds to one source token
  const int source_token = blockIdx.x;
  const int tid = threadIdx.x;

  // load prob into shared memory for the token
  // let first topk thread to load probs
  for (int i = tid; i < topk; i += blockDim.x * blockDim.y) {
    s_probs[i] = probs[source_token * topk + i];
  }
  __syncthreads();

  // TODO: use float for accumulator
  Fragment frag_sum;
  Fragment frag;

  for (int i = tid * kFragSize; i < dim; i += blockDim.x * kFragSize) {
    frag_sum.clear();

    // sum over topk
    for (int k = 0; k < topk; k++) {
      const int source_row = row_id_map[k * n_tokens + source_token];
      const T* source_row_ptr = permuted_tokens + source_row * dim;
      // load chunk from permuted tokens
      cutlass::arch::global_load<Fragment,
                                 sizeof(Fragment),
                                 cutlass::arch::CacheOperation::LastUse>(
          frag, (source_row_ptr + i), true);

      // apply probs
      frag = frag * s_probs[k];

      // sum
      for (int d = 0; d < kFragSize; d++) {
        frag_sum.at(d) = frag_sum.at(d) + frag.at(d);
      }
    }

    // store back to tokens
    T* dest_row_ptr = tokens + source_token * dim;
    *(float4*)(dest_row_ptr + i) = *(float4*)(frag_sum.data());
  }
}

template <typename T>
void launch_permute_kernel(
    const T* tokens,           // [n_tokens, dim]
    T* permuted_tokens,        // [n_permuted_tokens, dim]
    const int* sorted_row_id,  // [n_permuted_tokens] -> flattened index
    int* row_id_map,           // [topk, n_tokens] -> idx in permuted tokens
    const int n_tokens,
    const int topk,
    const int dim,
    hipStream_t stream) {
  const int n_permuted_tokens = n_tokens * topk;
  int threads = 256;
  int blocks = (n_permuted_tokens + threads - 1) / threads;
  permute_row_id_map<<<blocks, threads, 0, stream>>>(
      sorted_row_id, row_id_map, n_tokens, topk);

  // use 128-bit load/store
  constexpr int kFragSize = 16 / sizeof(T);
  // assert(dim % kFragSize == 0);

  // one block per source token
  blocks = n_tokens;
  threads = std::min(dim / kFragSize, 1024);
  // assert(topk <= 128);
  permute_kernel<T, kFragSize, /*TOPK=*/128><<<blocks, threads, 0, stream>>>(
      tokens, permuted_tokens, row_id_map, n_tokens, topk, dim);
}

template <typename T>
void launch_unpermute_kernel(
    const T* permuted_tokens,  // [n_permuted_tokens, dim]
    T* tokens,                 // [n_tokens, dim]
    int* row_id_map,           // [topk, n_tokens] => dst row
    const T* prob,             // [n_tokens, topk]
    const int n_tokens,
    const int topk,
    const int dim,
    hipStream_t stream) {
  // use 128-bit load/store
  constexpr int kFragSize = 16 / sizeof(T);
  // assert(dim % kFragSize == 0);

  // each block corresponds to one token
  int blocks = n_tokens;
  // up to 1024 threads per block
  int threads = std::min(dim / kFragSize, 1024);
  size_t smem_bytes = topk * sizeof(T);

  // unpermute_topK fwd
  unpermute_kernel<T, kFragSize><<<blocks, threads, smem_bytes, stream>>>(
      permuted_tokens, tokens, row_id_map, prob, n_tokens, topk, dim);
}

}  // namespace

std::tuple<torch::Tensor, torch::Tensor> permute(
    torch::Tensor tokens,  // [n_tokens, dim]
    torch::Tensor indices  // [n_tokens, topk]
) {
  const auto n_tokens = tokens.size(0);
  const auto dim = tokens.size(1);
  const auto topk = indices.size(1);

  const auto n_permuted_tokens = n_tokens * topk;
  const auto options = tokens.options();

  // calculate the size of temporary storage
  size_t temp_storage_bytes = 0;
  int* temp_ptr = nullptr;
  hipcub::DeviceRadixSort::SortPairs(nullptr,
                                  temp_storage_bytes,
                                  temp_ptr,
                                  temp_ptr,
                                  temp_ptr,
                                  temp_ptr,
                                  n_permuted_tokens);
  auto temp_storage =
      torch::empty(temp_storage_bytes, options.dtype(torch::kInt8));

  const auto int32_options = options.dtype(torch::kInt32);
  auto sorted_indices = torch::zeros(n_permuted_tokens, int32_options);
  auto row_id = torch::range(0, n_permuted_tokens - 1, 1, int32_options);
  auto sorted_row_id = torch::zeros(n_permuted_tokens, int32_options);

  const int* indices_ptr = indices.const_data_ptr<int>();
  const int* row_id_ptr = row_id.const_data_ptr<int>();
  int* sorted_indices_ptr = sorted_indices.data_ptr<int>();
  int* sorted_row_id_ptr = sorted_row_id.data_ptr<int>();
  void* d_temp_storage = temp_storage.data_ptr();

  // size_t temp_storage_bytes = std::numeric_limits<size_t>::max();
  hipcub::DeviceRadixSort::SortPairs(d_temp_storage,
                                  temp_storage_bytes,
                                  indices_ptr,
                                  sorted_indices_ptr,
                                  row_id_ptr,
                                  sorted_row_id_ptr,
                                  n_tokens * topk);

  const auto type = tokens.scalar_type();

  auto permuted_tokens = torch::empty({n_permuted_tokens, dim},
                                      torch::dtype(type).device(torch::kCUDA));
  auto row_id_map = torch::empty(
      {n_tokens * topk}, torch::dtype(torch::kInt32).device(torch::kCUDA));

  auto* stream = at::cuda::getCurrentCUDAStream().stream();

#define LAUNCH_PERMUTE_KERNEL(DType)                            \
  launch_permute_kernel<DType>(get_ptr<DType>(tokens),          \
                               get_ptr<DType>(permuted_tokens), \
                               sorted_row_id_ptr,               \
                               row_id_map.data_ptr<int>(),      \
                               n_tokens,                        \
                               topk,                            \
                               dim,                             \
                               stream);

  switch (type) {
    case torch::ScalarType::Float: {
      LAUNCH_PERMUTE_KERNEL(float);
      break;
    }
    case torch::ScalarType::Half: {
      LAUNCH_PERMUTE_KERNEL(cutlass::half_t);
      break;
    }
    case torch::ScalarType::BFloat16: {
      LAUNCH_PERMUTE_KERNEL(cutlass::bfloat16_t);
      break;
    }
    default:
      CHECK(false) << "Unsupported tensor type: " << type;
  }

  return {permuted_tokens, row_id_map};
}

torch::Tensor unpermute(
    torch::Tensor permuted_tokens,  // [n_permuted_tokens, dim]
    torch::Tensor row_id_map,       // [topk, n_tokens] => dst row
    torch::Tensor probs,            // [n_tokens, topk]
    int64_t n_tokens,
    int64_t topk) {
  const auto dim = permuted_tokens.size(1);
  const auto type = permuted_tokens.scalar_type();

  // [n_tokens, dim]
  auto tokens = torch::empty(
      {n_tokens, dim},
      torch::dtype(type).device(torch::kCUDA).requires_grad(false));

  auto* stream = at::cuda::getCurrentCUDAStream().stream();

#define LAUNCH_UNPERMUTE_KERNEL(DType)                            \
  launch_unpermute_kernel<DType>(get_ptr<DType>(permuted_tokens), \
                                 get_ptr<DType>(tokens),          \
                                 row_id_map.data_ptr<int>(),      \
                                 get_ptr<DType>(probs),           \
                                 n_tokens,                        \
                                 topk,                            \
                                 dim,                             \
                                 stream);

  switch (type) {
    case torch::ScalarType::Float: {
      LAUNCH_UNPERMUTE_KERNEL(float);
      break;
    }
    case torch::ScalarType::Half: {
      LAUNCH_UNPERMUTE_KERNEL(cutlass::half_t);
      break;
    }
    case torch::ScalarType::BFloat16: {
      LAUNCH_UNPERMUTE_KERNEL(cutlass::bfloat16_t);
      break;
    }
    default:
      CHECK(false) << "Unsupported tensor type: " << type;
  }

  return tokens;
}

}  // namespace llm::kernel::moe
