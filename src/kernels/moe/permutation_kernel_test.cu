#include "hip/hip_runtime.h"
#include <ATen/ops/allclose.h>
#include <c10/core/ScalarType.h>
#include <gtest/gtest.h>
#include <torch/torch.h>

#include <cute/layout.hpp>

#include "gtest/gtest.h"

namespace llm {

namespace kernel::moe {
// forward declare the kernel function
std::tuple<torch::Tensor, torch::Tensor> permute_with_index_map(
    torch::Tensor tokens,  // [n_tokens, dim]
    torch::Tensor indices  // [n_tokens, topk]
);

torch::Tensor unpermute_with_index_map(
    torch::Tensor permuted_tokens,  // [n_permuted_tokens, dim]
    torch::Tensor row_id_map,       // [topk, n_tokens]
    torch::Tensor probs             // [n_tokens, topk]
);

std::tuple<torch::Tensor, torch::Tensor> permute_with_mask_map(
    torch::Tensor tokens,       // [n_tokens, dim]
    torch::Tensor routing_map,  // [n_tokens, n_experts]
    int64_t topk);

torch::Tensor unpermute_with_mask_map(
    torch::Tensor permuted_tokens,  // [n_permuted_tokens, dim]
    torch::Tensor row_id_map,       // [n_experts, n_tokens]
    torch::Tensor probs             // [n_tokens, n_experts]
);

}  // namespace kernel::moe

namespace {
// reference implementation
std::tuple<torch::Tensor, torch::Tensor> permute_index_ref(
    const torch::Tensor& tokens,       // [n_tokens, dim]
    const torch::Tensor& topk_indices  // [n_tokens, topk]
) {
  const auto n_tokens = tokens.size(0);
  const auto topk = topk_indices.size(1);

  auto flatten_indices = topk_indices.view({-1});
  // idx, sorted by (experts, tokens)
  auto sorted_incices = flatten_indices.argsort(/*stable=*/true);

  // idx => token_indices, [n_permuted_tokens]
  auto token_indices = sorted_incices.div(topk, /*rounding_mode=*/"floor");
  auto permuted_tokens = tokens.index_select(
      /*dim=*/0, token_indices);

  return {permuted_tokens, sorted_incices};
}

torch::Tensor unpermute_index_ref(
    const torch::Tensor& permuted_tokens,  // [n_permuted_tokens, dim]
    const torch::Tensor& sorted_incices,   // [n_permuted_tokens]
    const torch::Tensor& probs,            // [n_token, topk]
    int64_t n_tokens,
    int64_t topK) {
  auto tokens = torch::zeros_like(permuted_tokens);

  // [n_permuted_tokens, dim] restore back to original order, sorted by (tokens)
  tokens.index_copy_(
      /*dim=*/0, sorted_incices, permuted_tokens);
  // [n_permuted_tokens, dim] => [n_tokens, topk, dim]
  tokens = tokens.reshape({n_tokens, topK, -1});

  // apply prob
  // [n_tokens, topk, dim] * [n_tokens, topk]
  tokens *= probs.unsqueeze(/*dim=*/-1);

  // [n_tokens, dim], sum over topk
  return tokens.sum(/*dim=*/1);
}

std::tuple<torch::Tensor, torch::Tensor> permute_mask_ref(
    const torch::Tensor& tokens,      // [n_tokens, dim]
    const torch::Tensor& routing_map  // [n_tokens, n_experts]
) {
  const auto n_tokens = routing_map.size(0);
  const auto n_experts = routing_map.size(1);
  const auto options = tokens.options();

  // [n_experts, n_tokens]
  auto token_indices = torch::arange(n_tokens, options.dtype(torch::kLong))
                           .unsqueeze(/*dim=*/0)
                           .expand({n_experts, n_tokens});

  // [n_permuted_tokens] original token indices, sorted by expert idx
  auto sorted_indices = token_indices.masked_select(/*mask=*/routing_map.t());
  auto permuted_tokens = tokens.index_select(
      /*dim=*/0, /*index=*/sorted_indices);
  return {permuted_tokens, sorted_indices};
}

torch::Tensor unpermute_mask_ref(
    const torch::Tensor& permuted_tokens,  // [n_permuted_tokens, dim]
    const torch::Tensor& permuted_probs,   // [n_permuted_tokens]
    const torch::Tensor& sorted_incices,   // [n_permuted_tokens]
    int64_t n_tokens) {
  const auto dim = permuted_tokens.size(1);
  const auto options = permuted_tokens.options();
  // [n_tokens, dim]
  auto tokens = torch::zeros({n_tokens, dim}, options);
  // [n_permuted_tokens] => [n_permuted_tokens, dim]
  auto index = sorted_incices.unsqueeze(/*dim=*/1).expand({-1, dim});
  // reduce sum over experts
  tokens.scatter_add_(
      /*dim=*/0,
      /*index=*/index,
      /*src=*/permuted_tokens * permuted_probs.unsqueeze(/*dim=*/1));
  return tokens;
}

}  // namespace

class PermuteTest
    : public ::testing::TestWithParam<std::tuple<torch::ScalarType /*q_dtype*/,
                                                 int64_t /*n_tokens*/,
                                                 int64_t /*dim*/,
                                                 int64_t /*n_experts*/,
                                                 int64_t /*topk*/>> {
 public:
  void SetUp() override {
    // Set random seed for test stability
    torch::manual_seed(0);
  }
};

TEST_P(PermuteTest, Index) {
  const auto [dtype, n_tokens, dim, n_experts, topk] = GetParam();

  const auto options = torch::dtype(dtype).device(torch::kCUDA);

  const auto tokens = torch::randn({n_tokens, dim}, options);
  const auto gating_logit = torch::randn({n_tokens, n_experts}, options);

  auto [weights, indices] = gating_logit.topk(topk, /*dim=*/-1);
  auto probs = weights.softmax(/*dim=*/-1);

  auto [permuted_tokens, sorted_indices] =
      kernel::moe::permute_with_index_map(tokens, indices.to(torch::kInt32));

  auto [ref_permuted_tokens, ref_sorted_indices] =
      permute_index_ref(tokens, indices);

  EXPECT_TRUE(torch::allclose(permuted_tokens, ref_permuted_tokens));

  auto unpermute_out = kernel::moe::unpermute_with_index_map(
      permuted_tokens, sorted_indices, probs);

  auto ref_unpermute_out = unpermute_index_ref(
      ref_permuted_tokens, ref_sorted_indices, probs, n_tokens, topk);
  EXPECT_TRUE(torch::allclose(
      unpermute_out, ref_unpermute_out, /*rtol=*/1e-2, /*atol=*/1e-2));
  EXPECT_TRUE(
      torch::allclose(tokens, unpermute_out, /*rtol=*/1e-2, /*atol=*/1e-2));
}

TEST_P(PermuteTest, Mask) {
  const auto [dtype, n_tokens, dim, n_experts, topk] = GetParam();

  const auto options = torch::dtype(dtype).device(torch::kCUDA);

  const auto tokens = torch::randn({n_tokens, dim}, options);
  const auto gating_logit = torch::randn({n_tokens, n_experts}, options);

  auto [weights, indices] = gating_logit.topk(topk, /*dim=*/-1);
  //   auto probs = weights.softmax(/*dim=*/-1);

  // construct dense routing map and probs
  auto probs = torch::zeros_like(gating_logit)
                   .scatter(
                       /*dim=*/1, /*index=*/indices, /*value=*/1.0 / topk);
  auto routing_map = torch::zeros_like(gating_logit, torch::kInt)
                         .scatter(
                             /*dim=*/1, /*index=*/indices, /*value=*/1)
                         .to(torch::kBool);

  auto [permuted_tokens, row_id_map] =
      kernel::moe::permute_with_mask_map(tokens, routing_map, topk);

  auto [ref_permuted_tokens, ref_row_id_map] =
      permute_mask_ref(tokens, routing_map);

  EXPECT_TRUE(torch::allclose(permuted_tokens, ref_permuted_tokens));

  auto unpermute_out =
      kernel::moe::unpermute_with_mask_map(permuted_tokens, row_id_map, probs);

  auto ref_permuted_probs = probs.t().masked_select(/*mask=*/routing_map.t());
  auto ref_unpermute_out = unpermute_mask_ref(
      ref_permuted_tokens, ref_permuted_probs, ref_row_id_map, n_tokens);
  EXPECT_TRUE(torch::allclose(
      unpermute_out, ref_unpermute_out, /*rtol=*/1e-2, /*atol=*/1e-2));

  EXPECT_TRUE(torch::allclose(tokens,
                              unpermute_out,
                              /*rtol=*/1e-2,
                              /*atol=*/1e-2));
}

INSTANTIATE_TEST_SUITE_P(
    SM80,
    PermuteTest,
    ::testing::Combine(::testing::Values(torch::kFloat,
                                         torch::kHalf,
                                         torch::kBFloat16),  // dtype
                       ::testing::Values(1, 2, 16),          // n_tokens
                       ::testing::Values(16, 64),            // dim
                       ::testing::Values(4, 8, 16),          // n_experts
                       ::testing::Values(1, 2, 4)            // topk
                       ));

}  // namespace llm
