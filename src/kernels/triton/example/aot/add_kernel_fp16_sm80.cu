/* clang-format off */
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <assert.h>

// launcher for: add_kernel_fp16_sm80_16_warps1xstages3
hipError_t add_kernel_fp16_sm80_6a55b24f_0123(hipStream_t stream, hipDeviceptr_t x_ptr, hipDeviceptr_t y_ptr, hipDeviceptr_t output_ptr, int32_t n_elements);

hipError_t add_kernel_fp16_sm80_16_warps1xstages3(hipStream_t stream, hipDeviceptr_t x_ptr, hipDeviceptr_t y_ptr, hipDeviceptr_t output_ptr, int32_t n_elements){
if (1)
    return add_kernel_fp16_sm80_6a55b24f_0123(stream, x_ptr, y_ptr, output_ptr, n_elements);

  return hipErrorInvalidValue;
}

// load for: add_kernel_fp16_sm80_16_warps1xstages3
void load_add_kernel_fp16_sm80_6a55b24f_0123();
void load_add_kernel_fp16_sm80_16_warps1xstages3() {
  load_add_kernel_fp16_sm80_6a55b24f_0123();
}

// unload for: add_kernel_fp16_sm80_16_warps1xstages3
void unload_add_kernel_fp16_sm80_6a55b24f_0123();
void unload_add_kernel_fp16_sm80_16_warps1xstages3() {
  unload_add_kernel_fp16_sm80_6a55b24f_0123();
}

typedef hipError_t (*kernel_func_t)(hipStream_t stream, hipDeviceptr_t x_ptr, hipDeviceptr_t y_ptr, hipDeviceptr_t output_ptr, int32_t n_elements);
kernel_func_t add_kernel_fp16_sm80_kernels[] = {
  add_kernel_fp16_sm80_16_warps1xstages3,
};

int add_kernel_fp16_sm80_get_num_algos(void){
  return (int)(sizeof(add_kernel_fp16_sm80_kernels) / sizeof(add_kernel_fp16_sm80_kernels[0]));
}

hipError_t add_kernel_fp16_sm80(hipStream_t stream, hipDeviceptr_t x_ptr, hipDeviceptr_t y_ptr, hipDeviceptr_t output_ptr, int32_t n_elements, int algo_id){
  assert (algo_id < (int)sizeof(add_kernel_fp16_sm80_kernels));
  return add_kernel_fp16_sm80_kernels[algo_id](stream, x_ptr, y_ptr, output_ptr, n_elements);
}

void load_add_kernel_fp16_sm80(void){
  load_add_kernel_fp16_sm80_16_warps1xstages3();
}

void unload_add_kernel_fp16_sm80(void){
  unload_add_kernel_fp16_sm80_16_warps1xstages3();
}


hipError_t add_kernel_fp16_sm80_default(hipStream_t stream, hipDeviceptr_t x_ptr, hipDeviceptr_t y_ptr, hipDeviceptr_t output_ptr, int32_t n_elements){
  return add_kernel_fp16_sm80(stream, x_ptr, y_ptr, output_ptr, n_elements, 0);
}
