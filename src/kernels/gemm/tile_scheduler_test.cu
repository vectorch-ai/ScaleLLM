#include "hip/hip_runtime.h"
#include <gtest/gtest.h>

#include <cute/tensor.hpp>

#include "tile_scheduler.cuh"

namespace llm {

class TileSchedulerTest
    : public ::testing::TestWithParam<std::tuple<int32_t /*cluster_m*/,
                                                 int32_t /*cluster_n*/,
                                                 int32_t /*grid_m*/,
                                                 int32_t /*grid_n*/,
                                                 int32_t /*swizzle*/,
                                                 RasterOrder /*order*/>> {};

// StaticPersistentTileScheduler
TEST_P(TileSchedulerTest, StaticPersistent) {
  using TileScheduler = StaticPersistentTileScheduler;
  using namespace cute;

  const auto [cluster_m, cluster_n, grid_m, grid_n, swizzle, order] =
      GetParam();

  TileScheduler::Params params{
      cluster_m, cluster_n, grid_m, grid_n, swizzle, order};

  const int problem_tiles = params.grid_shape_m * params.grid_shape_n;
  // std::vector<int> mapping_data(problem_tiles);
  // auto mapping =
  //     make_tensor(mapping_data.data(),
  //                 make_shape(params.grid_shape_m, params.grid_shape_n));
  int pre_tile_m = 0, pre_tile_n = 0;
  const int max_dist = order == RasterOrder::AlongM
                           ? (swizzle * cluster_n) + cluster_m
                           : (swizzle * cluster_m) + cluster_n;
  int32_t valid = 0;
  for (int linear_idx = 0; linear_idx < problem_tiles; ++linear_idx) {
    const auto [tile_m, tile_n] =
        TileScheduler::swizzle_and_rasterize(linear_idx, params);

    const int dist =
        std::abs(tile_m - pre_tile_m) + std::abs(tile_n - pre_tile_n);
    pre_tile_m = tile_m;
    pre_tile_n = tile_n;
    EXPECT_LE(dist, max_dist);
    // mapping(tile_m, tile_n) = linear_idx;

    // (grid_m, grid_n):(1, grid_m)
    const int idx = tile_m + (tile_n * grid_m);
    valid ^= idx;
    valid ^= linear_idx;
  }
  EXPECT_EQ(valid, 0);

  // print_tensor(mapping);
}

INSTANTIATE_TEST_SUITE_P(
    TileScheduler,
    TileSchedulerTest,
    ::testing::Combine(::testing::Values(1, 2),     // cluster_m
                       ::testing::Values(1, 2),     // cluster_n
                       ::testing::Values(8, 16),    // grid_m
                       ::testing::Values(8, 16),    // grid_n
                       ::testing::Values(1, 2, 4),  // swizzle
                       ::testing::Values(RasterOrder::AlongM,
                                         RasterOrder::AlongN)  // order
                       ));

}  // namespace llm
